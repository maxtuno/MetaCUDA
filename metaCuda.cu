/**
    Research 4 Fun

    metaCuda.cu

    Purpose: Calculates the n-th Fibonacci number an the Factorial of a number
    from CUDA + Template Meta-Programming

    @author O. A. Riveros
    @version 1.0 28 May 2014 Santiago Chile.
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
using namespace std;

// Begin CUDA

///////////////
// Fibonacci //
///////////////
template<unsigned long N>
__device__ unsigned long  cuMetaFibonacci() {
	return  cuMetaFibonacci<N - 1>() + cuMetaFibonacci<N - 2>();
}

template<>
__device__ unsigned long  cuMetaFibonacci<0>() {
	return 1;
}

template<>
__device__ unsigned long  cuMetaFibonacci<1>() {
	return 1;
}

template<>
__device__ unsigned long  cuMetaFibonacci<2>() {
	return 1;
}

template<unsigned long N>
__global__ void cuFibonacci(unsigned long *out) {
	*out = cuMetaFibonacci<N>();
}

///////////////
// Factorial //
///////////////
template<unsigned long N>
__device__ unsigned long  cuMetaFactorial() {
	return  N * cuMetaFactorial<N - 1>();
}

template<>
__device__ unsigned long  cuMetaFactorial<1>() {
	return 1;
}

template<unsigned long N>
__global__ void cuFactorial(unsigned long *out) {
	*out = cuMetaFactorial<N>();
}

// End CUDA

int main() {

	///////////////
	// Fibonacci //
	///////////////

	size_t size = sizeof(unsigned long);

	unsigned long h_out[] = { 0 };
	unsigned long *d_out;

	hipMalloc((void **) &d_out, size);
	hipMemcpy(d_out, h_out, size, hipMemcpyHostToDevice);

	clock_t startTime = clock();

	cuFibonacci<20> <<<1, 1>>>(d_out);

	clock_t endTime = clock();
	clock_t clockTicksTaken = endTime - startTime;

	hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

	cout << h_out[0] << endl;

	hipFree(d_out);

	double timeInSeconds = clockTicksTaken / (double) CLOCKS_PER_SEC;
	cout << timeInSeconds << endl;

	///////////////
	// Factorial //
	///////////////

	hipMalloc((void **) &d_out, size);
	hipMemcpy(d_out, h_out, size, hipMemcpyHostToDevice);

	startTime = clock();

	cuFactorial<20> <<<1, 1>>>(d_out);

	endTime = clock();
	clockTicksTaken = endTime - startTime;

	hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

	cout << h_out[0] << endl;

	hipFree(d_out);

	timeInSeconds = clockTicksTaken / (double) CLOCKS_PER_SEC;
	cout << timeInSeconds << endl;

}

// Original Output
// 11:56:05 Build Finished (took 16s.185ms)
// 6765
// 4.2e-05
// 2432902008176640000
// 9e-06
